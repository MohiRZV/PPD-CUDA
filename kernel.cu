#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/opencv.hpp>
using namespace std;


#define THREADS_NO 512

hipError_t laplaceFilterWithCuda(unsigned char* matrix, unsigned char* result_matrix, float* filter, int rows, int columns);

__global__ void laplaceFilterKernel(unsigned char* matrix, unsigned char* result, float* filter, int rows, int columns, int filter_rows, int filter_columns) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < rows * columns) {
        // get row and column index from liniarised matrix index
        int row = index / columns;
        int column = index % columns;

        // compute limits
        int upperLimit = row - filter_rows / 2;
        int lowerLimit = row + filter_rows / 2;
        int leftLimit = column - filter_columns / 2;
        int rightLimit = column + filter_columns / 2;

        int filterStartLine = 0;
        int filterEndLine = filter_rows;
        int filterStartColumn = 0;
        int filterEndColumn = filter_columns;

        // upper limit overflow
        if (upperLimit < 0) {
            filterStartLine = 0 - upperLimit;
            upperLimit = 0;
        }
        // lower limit overflow
        if (lowerLimit > rows - 1) {
            filterEndLine = filter_rows - (lowerLimit - rows + 1);
            lowerLimit = rows - 1;
        }
        // left limit overflow
        if (leftLimit < 0) {
            filterStartColumn = 0 - leftLimit;
            leftLimit = 0;
        }
        // right limit overflow
        if (rightLimit > columns - 1) {
            filterEndColumn = filter_columns - (rightLimit - columns + 1);
            rightLimit = columns - 1;
        }

        int filterLine = filterStartLine, filterColumn = filterStartColumn;
        float sum = 0;
        // overlap kernel over current element
        for (int i = upperLimit; i <= lowerLimit; i++) {
            for (int j = leftLimit; j <= rightLimit; j++) {
                sum += matrix[i * columns + j] * filter[filterLine * filter_columns + filterColumn];
                filterColumn++;
            }
            filterLine++;
            filterColumn = filterStartColumn;
        }
        // save computed result
        result[row * rows + column] = (char)sum / (filter_columns * filter_rows);
    }

}

// Laplacian filter - for determining edges
void filter_laplacian_init(float* filter) {
    filter[0] = 0;
    filter[1] = -1;
    filter[2] = 0;
    filter[3] = -1;
    filter[4] = 4;
    filter[5] = -1;
    filter[6] = 0;
    filter[7] = -1;
    filter[8] = 0;
}

void filter_blur_init(float* filter) {
    //float factor = 1;
    //filter[0] = factor * 0;
    //filter[1] = factor * -1;
    //filter[2] = factor * 0;
    //filter[3] = factor * -1;
    //filter[4] = factor * 5;
    //filter[5] = factor * -1;
    //filter[6] = factor * 0;
    //filter[7] = factor * -1;
    //filter[8] = factor * 0;
    float factor = 1;
    filter[0] = factor * 1;
    filter[1] = factor * 2;
    filter[2] = factor * 1;
    filter[3] = factor * 2;
    filter[4] = factor * 4;
    filter[5] = factor * 2;
    filter[6] = factor * 1;
    filter[7] = factor * 2;
    filter[8] = factor * 1;
}

int main()
{
    // declare input and output openCV matrixes
    cv::Mat inputImageRGBA;
    cv::Mat outputImageRGBA;

    // declare input and output values matrixes
    uchar4* inputImageRGBAMatrix;
    uchar4* outputImageRGBAMatrix;

    // input, output files
    string input_file{ "image.jpg" };
    string output_file{ "output.jpg" };
    string image_path = cv::samples::findFile(input_file);
    cv::cvtColor(cv::imread(image_path, 1), inputImageRGBA, 2);

    if (inputImageRGBA.empty()) {
        std::cerr << "Couldn't open file: " << input_file << std::endl;
        exit(1);
    }

    int numRows = inputImageRGBA.rows;
    int numCols = inputImageRGBA.cols;

    inputImageRGBAMatrix = new uchar4[numRows * numCols];
    outputImageRGBAMatrix = new uchar4[numRows * numCols];

    outputImageRGBA.create(numRows, numCols, CV_8UC4);

    memcpy(inputImageRGBAMatrix, (uchar4*)inputImageRGBA.ptr<unsigned char>(0), numRows * numCols * sizeof(uchar4));

    const size_t numPixels = numRows * numCols;

    unsigned char* grayscale = new unsigned char[numPixels];

    unsigned char* laplacian = new unsigned char[numPixels];

    float* filter = new float[3 * 3];

    for (size_t i = 0; i < numRows * numCols; ++i) {
        uchar4 rgba = inputImageRGBAMatrix[i];
        // convert to grayscale
        grayscale[i] = 0.2989 * rgba.x + 0.5870 * rgba.y + 0.1140 * rgba.z;
    }

    filter_laplacian_init(filter);
    hipError_t cudaStatus = laplaceFilterWithCuda(grayscale, laplacian, filter, numRows, numCols);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "laplaceFilterWithCuda failed!");
        return 1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    for (size_t i = 0; i < numRows * numCols; ++i) {
        uchar4 rgba = make_uchar4(laplacian[i], laplacian[i], laplacian[i], 255);
        outputImageRGBAMatrix[i] = rgba;
    }

    memcpy((uchar4*)outputImageRGBA.ptr<unsigned char>(0), outputImageRGBAMatrix, numRows * numCols * sizeof(uchar4));

    cv::Mat imageOutputRGB;
    cv::cvtColor(outputImageRGBA, imageOutputRGB, 3);
    cv::imwrite(output_file.c_str(), imageOutputRGB);

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    delete[] grayscale;
    delete[] laplacian;

    return 0;
}

hipError_t laplaceFilterWithCuda(unsigned char* matrix, unsigned char* result_matrix, float* filter, int rows, int columns)
{
    unsigned char* dev_matrix = 0;
    unsigned char* dev_result = 0;
    float* dev_filter = 0;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_result, rows * columns * sizeof(unsigned char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_matrix, rows * columns * sizeof(unsigned char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_filter, 3 * 3 * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_matrix, matrix, rows * columns * sizeof(unsigned char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_result, result_matrix, rows * columns * sizeof(unsigned char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_filter, filter, 3 * 3 * sizeof(float), hipMemcpyHostToDevice);

    int blocksNo = rows * columns / (THREADS_NO - 1);
    laplaceFilterKernel << <blocksNo, THREADS_NO >> > (dev_matrix, dev_result, dev_filter, rows, columns, 3, 3);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "laplaceFilterKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching laplaceFilterKernel!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(result_matrix, dev_result, rows * columns * sizeof(unsigned char), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_matrix);
    hipFree(dev_result);
    hipFree(dev_filter);

    return cudaStatus;

}
